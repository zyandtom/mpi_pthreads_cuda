#include "hip/hip_runtime.h"
/* 
 *ZENG Yang
 *20711899
 *yzengav@connect.ust.hk

 * COMPILE: nvcc -std=c++11 clustering_cuda_skeleton.cu clustering_impl.cpp main.cpp -o cuda
 * RUN:     ./cuda <path> <epsilon> <mu> <num_blocks_per_grid> <num_threads_per_block>
 */

#include <iostream>
#include "clustering.h"

// Define variables or functions here
__global__ void kernel(int *d_nbrs, int *d_nbr_offs, bool *d_pivots, int *d_num_sim_nbrs, int *d_sim_nbrs,
    int num_vs, float epsilon, int mu) {
    const int tid = blockDim.x*blockIdx.x + threadIdx.x;
    const int nthread = blockDim.x*gridDim.x;
    //printf("nthread is %d\n", nthread);
    
    //stage 1
    for(int i = tid; i < num_vs; i += nthread) {
        //printf("this is the %dth num_vs\n", i);
        int left_start = d_nbr_offs[i];
        //printf("d_nbr_offs is  %d\n", d_nbr_offs[i]);
        int left_end = d_nbr_offs[i + 1];
        int left_size = left_end - left_start;
        //printf("left_size is %d\n", left_size);

        int cur_pos = d_nbr_offs[i];
        // loop over all neighbors of i
        for (int j = left_start; j < left_end; j++) {
            int nbr_id = d_nbrs[j];

            int right_start = d_nbr_offs[nbr_id];
            int right_end = d_nbr_offs[nbr_id + 1];
            int right_size = right_end - right_start;

            // compute the similarity
            int left_pos = left_start, right_pos = right_start, num_com_nbrs = 0;
        
            while (left_pos < left_end && right_pos < right_end) {
                if (d_nbrs[left_pos] == d_nbrs[right_pos]) {
                    num_com_nbrs++;
                    left_pos++;
                    right_pos++;
                } else if (d_nbrs[left_pos] < d_nbrs[right_pos]) {
                    left_pos++;
                } else {
                    right_pos++;
                }
            }
            
            float sim = (num_com_nbrs + 2) / std::sqrt((left_size + 1.0) * (right_size + 1.0));

            if (sim > epsilon) {
                d_sim_nbrs[cur_pos + d_num_sim_nbrs[i]] = nbr_id;
                d_num_sim_nbrs[i]++;
            }
        }
        //printf("compute sim done!\n");
        if (d_num_sim_nbrs[i] > mu){
            d_pivots[i] = true;
        }
    }
/*
    for (int i = 0; i < num_vs; i++){
        printf("pivot is %d", d_pivots[i]);
    }
*/
}

void expansion(int cur_id, int num_clusters, int *num_sim_nbrs, int *sim_nbrs,
               bool *visited, bool *pivots, int *cluster_result, int *nbr_offs) {
  for (int i = 0; i < num_sim_nbrs[cur_id]; i++) {
    int nbr_id = sim_nbrs[nbr_offs[cur_id] + i];
    if ((pivots[nbr_id])&&(!visited[nbr_id])){
      visited[nbr_id] = true;
      cluster_result[nbr_id] = num_clusters;
      expansion(nbr_id, num_clusters, num_sim_nbrs, sim_nbrs, visited, pivots,
                cluster_result, nbr_offs);
    }
  }
}

void cuda_scan(int num_vs, int num_es, int *nbr_offs, int *nbrs,
        float epsilon, int mu, int num_blocks_per_grid, int num_threads_per_block,
        int &num_clusters, int *cluster_result) {

    // Fill in the cuda_scan function here
    //printf("num_vs is %d\n", num_vs);

    bool *h_pivots;
    int *h_num_sim_nbrs;
    int *h_sim_nbrs;
    
    int *d_nbrs;
    int *d_nbr_offs;
    bool *d_pivots;
    int *d_num_sim_nbrs;
    int *d_sim_nbrs;
    
    size_t numvs_bool = num_vs * sizeof(bool);
    size_t numvs_int = num_vs * sizeof(int);
    //size_t size_sim_nbrs = num_vs * num_vs * sizeof(int);
    size_t nbrs_len = (num_es + 1) * sizeof(int);
    //size_t nbrs_len = sizeof(nbrs) / sizeof(int);
    //for (int i = 0; i < num_vs + 1; i++){
    //    printf("nbrs_offs is %d\n", nbr_offs[i]);
    //}
    //printf("nbrs_len is %d\n", nbrs_len);
    size_t nbr_offs_len = (num_vs + 1) * sizeof(int);
    //printf("nbrs_offs_len is %d\n", sizeof (nbr_offs));


    h_pivots = (bool *) malloc(numvs_bool);
    h_num_sim_nbrs = (int *) malloc(numvs_int);
    h_sim_nbrs = (int *) malloc(nbrs_len);

    memset(h_pivots, 0, numvs_bool);
    memset(h_num_sim_nbrs, 0, numvs_int);
    memset(h_sim_nbrs, 0, nbrs_len);
    //for (int i = 0; i < num_vs; i++){
    //    printf("h_num_sim_nbrs is %d\n", h_num_sim_nbrs[i]);
    //}
    
    hipMalloc(&d_nbrs, nbrs_len);
    hipMalloc(&d_nbr_offs, nbr_offs_len);
    hipMalloc(&d_pivots, numvs_bool);
    hipMalloc(&d_num_sim_nbrs, numvs_int);
    hipMalloc(&d_sim_nbrs, nbrs_len);

    hipMemcpy(d_nbrs, nbrs, nbrs_len, hipMemcpyHostToDevice);
    hipMemcpy(d_nbr_offs, nbr_offs, nbr_offs_len, hipMemcpyHostToDevice);
    hipMemcpy(d_pivots, h_pivots, numvs_bool, hipMemcpyHostToDevice);
    hipMemcpy(d_num_sim_nbrs, h_num_sim_nbrs, numvs_int, hipMemcpyHostToDevice);
    hipMemcpy(d_sim_nbrs, h_sim_nbrs, nbrs_len, hipMemcpyHostToDevice);

    kernel<<<num_blocks_per_grid, num_threads_per_block>>>(d_nbrs, d_nbr_offs, d_pivots, 
        d_num_sim_nbrs, d_sim_nbrs, num_vs, epsilon, mu);

    hipMemcpy(h_num_sim_nbrs, d_num_sim_nbrs, numvs_int, hipMemcpyDeviceToHost);
    hipMemcpy(h_sim_nbrs, d_sim_nbrs, nbrs_len, hipMemcpyDeviceToHost);
    hipMemcpy(h_pivots, d_pivots, numvs_bool, hipMemcpyDeviceToHost);
    
    //stage 2
    num_clusters = 0;
    bool *visited = new bool[num_vs]();

    for (int i = 0; i < num_vs; i++) {
        if (!h_pivots[i] || visited[i]) continue;

        visited[i] = true;
        cluster_result[i] = i;
        expansion(i, i, h_num_sim_nbrs, h_sim_nbrs, visited, h_pivots, cluster_result, nbr_offs);

        num_clusters ++;
    }
/*
    free(h_pivots);
    free(h_sim_nbrs);
    free(h_num_sim_nbrs);
    free(nbrs);
    free(nbr_offs);
    hipFree(d_nbrs);
    hipFree(d_nbr_offs);
    hipFree(d_pivots);
    hipFree(d_num_sim_nbrs);
    hipFree(d_sim_nbrs);
    */
}
